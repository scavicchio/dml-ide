#include "hip/hip_runtime.h"
#include "model.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

// CUDA ERROR CHECK WRAPPER
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort) {
            char buffer[200];
            snprintf(buffer, sizeof(buffer), "GPUassert error in CUDA kernel: %s %s %d\n", hipGetErrorString(code), file, line);
            std::string buffer_string = buffer;
            throw std::runtime_error(buffer_string);
        }
    }
}

/**__device__ bool insideTriangle(vec3 a, vec3 b, vec3 c, vec3 &point) {
    vec3 ca = c - a;
    vec3 ba = b - a;
    vec3 pa = point - a;

    float dotCACA = dot(ca, ca);
    float dotCABA = dot(ca, ba);
    float dotCAPA = dot(ca, pa);
    float dotBABA = dot(ba, ba);
    float dotBAPA = dot(ba, pa);

    float invDenom = 1.0f / (dotCACA * dotBABA - dotCABA * dotCABA);
    float u = (dotBABA * dotCAPA - dotCABA * dotBAPA) * invDenom;
    float v = (dotCACA * dotBAPA - dotCABA * dotCAPA) * invDenom;

    return (u >= 0.0f) && (v >= 0.0f) && (u + v < 1.0f);
}

__global__ void intersectPlane(int n_triangles, vec3 *model, vec3 point, vec3 dir, bool *d_intersects) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n_triangles) {

        vec3 &v1 = model[3*idx];
        vec3 &v2 = model[3*idx + 1];
        vec3 &v3 = model[3*idx + 2];
        bool &ret = d_intersects[idx];

        vec3 n = cross((v2 - v1), (v3 - v2));
        n = normalize(n);

        float dn = dot(dir, n);

        if (fabs(dn) < EPSILON) {
            ret = false;
        }

        float dist = -dot((point-v1), n)/dn;
        vec3 p = point + dir*dist;

        ret = insideTriangle(v1, v2, v3, p);
    }
}


bool simulation_data::isInsideCuda(glm::vec3 point, int n_model) {

    uint modelStart = 0;
    uint modelEnd = 0;
    int intersections = 0;

    if (n_model != 0)
        modelStart = model_indices[n_model-1];
    modelEnd = model_indices[n_model];

    glm::vec3 dir = Utils::randDirection();

    int n_tri = (modelEnd - modelStart) / 3;
    vec3 *d_vertices;
    vec3 *p;
    bool *intersects;

    //qDebug() << "Allocating for N triangles:" << n_tri;
    //qDebug() << "Mode start end" << modelStart << modelEnd;

    // Allocate cuda memory for results
    hipMallocManaged(&intersects, n_tri*sizeof(bool));

    int blockSize = 256;
    int numBlocks = (n_tri + blockSize - 1) / blockSize;
    intersectPlane<<<numBlocks, blockSize>>>(n_tri, this->d_vertices, point, dir, intersects);
    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();
    // Add up intersections after CUDA is done
    for (int i = 0; i < n_tri; i++) {
        if (intersects[i]) intersections++;
    }

    // Free memory
    hipFree(intersects);

    if (intersections % 2 == 1) {
        qDebug() << "FOUND INSIDE POINT";
        return true;
    } else
        return false;
}**/

void simulation_data::copyToGPU() {
    //float *h_vertices = new float[vertices.size() * 3];
    //int count = 0;
    //for (auto v : vertices) {
    //    h_vertices[count++] = v.x;
    //    h_vertices[count++] = v.y;
    //    h_vertices[count++] = v.z;
    //}

    float *CUDA_VERTICES;

    gpuErrchk(hipMalloc(&CUDA_VERTICES, sizeof(float)));
    gpuErrchk(hipMemcpy(CUDA_VERTICES, vertices.data(), sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipFree(CUDA_VERTICES));
}

/**
 * @brief Volume::Volume default constructor
 */
Volume::Volume() {

}

Volume::Volume(QString s_id) {
    id = s_id;
}

/**
 * @brief Volume::Volume full constructor
 * @param s_id ID
 * @param s_primitive Primitive type (e.g stl, obj, ...)
 * @param s_url File path for primitive mesh
 * @param s_units Units (e.g. mm)
 * @param s_rendering Rendering options
 * @param s_alpha Alpha value in [0.0,1.0]
 * @param s_color RGB string "r g b" where r,g,b in [0.0,1.0]
 */
Volume::Volume(QString s_id,
        QString s_primitive,
        QString s_url,
        QString s_units,
        QString s_rendering,
        QString s_alpha,
        QString s_color) {

    id = s_id;
    primitive = s_primitive;
    rendering = s_rendering;
    units = s_units;
    color = QVector4D(1.0f, 1.0f, 1.0f, 1.0f);

    url = QUrl(s_url);

    if (s_color != nullptr) {
        QList<QString> s_colors = s_color.split(" ");
        QList<float> f_colors;
        for (auto c : s_colors) {
            float f_c = c.toFloat();
            f_colors.append(f_c);
        }
        color.setX(f_colors.at(0));
        color.setY(f_colors.at(1));
        color.setZ(f_colors.at(2));
    }

    if (s_alpha != nullptr) {
        float f_alpha = s_alpha.toFloat();
        color.setW(f_alpha);
    }

    model = new model_data();
}

/**
 * @brief Design::Design default constructor
 */
Design::Design() {
    volumes = std::vector<Volume *>();
}
