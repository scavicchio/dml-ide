#include "hip/hip_runtime.h"
#include "simulatorParallel.h"


namespace cg = cooperative_groups;


// CUDA ERROR CHECK WRAPPER
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=false)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort) {
            char buffer[200];
            snprintf(buffer, sizeof(buffer), "GPUassert error in CUDA kernel: %s %s %d\n", hipGetErrorString(code), file, line);
            std::string buffer_string = buffer;
            throw std::runtime_error(buffer_string);
        }
    }
}

// --------------------------------------------------------------------
//  PARELLEL FUNCTIONS
// --------------------------------------------------------------------

//  Shared memory utility struct to avoid extern linker errors
//  See https://docs.nvidia.com/cuda/cuda-samples/index.html#cuda-parallel-reduction
// --------------------------------------------------------------------
template<class T> struct SharedMemory {
// --------------------------------------------------------------------
    __device__ inline operator T *() {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// Binary operator to compare two springs by stress
//__host__ __device__ CUDA_SPRING * springPtr_minStressOperator(CUDA_SPRING * &lhs, CUDA_SPRING * &rhs) {
//    return lhs->_max_stress < rhs->_max_stress ? lhs : rhs;
//}

// Reduction kernel templates with float data type and max operator
// See https://docs.nvidia.com/cuda/cuda-samples/index.html#cuda-parallel-reduction
// and Wilt, Nicolas "The Cuda Handbook" Listing 12.3
/**template<unsigned int blockSize>
__device__ void maxF_reduceLogStep(float *out, volatile float *partials) {
    const int tid = threadIdx.x;
    if (blockSize >= 1024) {
        if (blockSize < 512) {
            partials[tid] = std::max(partials[tid], partials[tid + 512]);
        }
        __syncThreads();
    }
    if (blockSize >= 512) {
        if (blockSize < 256) {
            partials[tid] = std::max(partials[tid], partials[tid + 256]);
        }
        __syncThreads();
    }
    if (blockSize >= 256) {
        if (blockSize < 128) {
            partials[tid] = std::max(partials[tid], partials[tid + 128]);
        }
        __syncThreads();
    }
    if (blockSize >= 128) {
        if (blockSize < 64) {
            partials[tid] = std::max(partials[tid], partials[tid + 64]);
        }
        __syncThreads();
    }

    if (tid < 32) {
        if (blockSize >= 64) partials[tid] = std::max(partials[tid], partials[tid + 32]);
        if (blockSize >= 32) partials[tid] = std::max(partials[tid], partials[tid + 16]);
        if (blockSize >= 16) partials[tid] = std::max(partials[tid], partials[tid + 8]);
        if (blockSize >= 8 ) partials[tid] = std::max(partials[tid], partials[tid + 4]);
        if (blockSize >= 4 ) partials[tid] = std::max(partials[tid], partials[tid + 2]);
        if (blockSize >= 2 ) partials[tid] = std::max(partials[tid], partials[tid + 1]);
        if (tid == 0) {
            *out = partials[0];
        }
    }
}**/

/**template<unsigned int blockSize>
__device__ void maxF_reducePass(float *out, float *partial, const float *in, unsigned int n) {

    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    float maxF = 0;

    for (size_t i = blockIdx.x*blockSize + tid; i < n; i += blockSize*gridDim.x) {
        maxF = std::max(maxF, in[i]);
    }
    sdata[tid] = maxF;
    __syncThreads();

    if (gridDim.x == 1) {
        maxF_reduceLogStep<blockSize>(&out[blockIdx.x], sdata);
        return;
    }
    maxF_reduceLogStep<blockSize>(&partial[blockIdx.x], sdata);

    __shared__ bool lastBlock;
    __threadFence();

    if (tid == 0) {
        unsigned int ticket = atomicAdd(&threadsDone, 1);
        lastBlock = (ticket == gridDim.x-1);
    }
    __syncThreads();

    if (lastBlock) {
        float lmaxF = 0;
        for (size_t i = tid; i < gridDim.x; i += blockSize) {
            lmaxF = std::max(lmaxF, partial[i];
        }
        sdata[threadIdx.x] = lmaxF;
        __syncThreads();
        maxF_reduceLogStep<blockSize>(out, partial);
        threadsDone = 0;
    }
}**/

// See https://docs.nvidia.com/cuda/cuda-samples/index.html#cuda-parallel-reduction
__global__ void warpReduce3MinSpring(CUDA_SPRING ** d_min_stress_out, CUDA_SPRING ** d_spring, int num_springs) {

    // Thread block group
    cg::thread_block cta = cg::this_thread_block();
    CUDA_SPRING ** s_spring = SharedMemory<CUDA_SPRING *>();

    // First level of reduction
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    CUDA_SPRING * result = (i < num_springs) ? d_spring[i] : nullptr;

    if (i + blockDim.x < num_springs) {
        result = result->_max_stress < d_spring[i+blockDim.x]->_max_stress ? result : d_spring[i+blockDim.x];
    }
    s_spring[tid] = result;
    cg::sync(cta);

    // Shared Memory Reduction
    for (unsigned int s = blockDim.x/2; s > 0; s>>=1) {
        if (tid < s) {
            s_spring[tid] = result = result->_max_stress < s_spring[tid + s]->_max_stress ? result : s_spring[tid + s];
        }
        cg::sync(cta);
    }

    if (tid == 0) d_min_stress_out[blockIdx.x] = result;
}


// CONSTRUCTOR
// --------------------------------------------------------------------
SimulatorParallel::SimulatorParallel(Simulation *sim, SimulationConfig *config)
// --------------------------------------------------------------------
{
    this->sim = sim;
    this->config = config;

    status = NOT_RUNNING;

    updates = 0;
    //sdkCreateTimer(&timer);
    //gpuErrchk(hipEventCreate(&startEvent));
    //gpuErrchk(hipEventCreate(&stopEvent));
    //gpuErrchk(hipEventCreate(&hostMemSyncEvent));
    removeLeastStressedBar();
}

SimulatorParallel::~SimulatorParallel()
{
    unregisterGraphicsResources();
}


// --------------------------------------------------------------------
// SIMULATION CONTROL
// --------------------------------------------------------------------

// TODO: add input
// Starts simulation timer and begins loop
void SimulatorParallel::start() {

    if (sim->running() || sim->masses.size() == 0) return;

    updates = 0;
    sdkStartTimer(&timer);
}

// Pauses simulation and syncs state back to host
// --------------------------------------------------------------------
void SimulatorParallel::pause() {
// --------------------------------------------------------------------

    if (!sim->running() || sim->masses.size() == 0) return;


}


// Steps simulation one timestep and syncs data back to host
// --------------------------------------------------------------------
void SimulatorParallel::step() {
// --------------------------------------------------------------------

    if (sim->running() || sim->masses.size() == 0) return;

    sim->step(sim->masses.front()->dt);

    hipEventSynchronize(hostMemSyncEvent);
    exportGraphicsVertices();
    exportGraphicsIndices();

    float milli = 1;
    gpuErrchk(hipEventRecord(stopEvent));
    gpuErrchk(hipEventSynchronize(stopEvent));
    gpuErrchk(hipEventElapsedTime(&milli, startEvent, stopEvent));
}


// Restarts simulation
// --------------------------------------------------------------------
void SimulatorParallel::stop() {
// --------------------------------------------------------------------

    sdkResetTimer(&timer);
}


// --------------------------------------------------------------------
// SIMULATION FUNCTIONS
// --------------------------------------------------------------------


// --------------------------------------------------------------------
void SimulatorParallel::removeLeastStressedBar() {
// --------------------------------------------------------------------

    hipDeviceSynchronize();
    uint numBlocks = sim->springBlocksPerGrid;

    CUDA_SPRING * leastStress = nullptr;
    CUDA_SPRING ** h_out = (CUDA_SPRING **) malloc(numBlocks*sizeof(CUDA_SPRING *));
    CUDA_SPRING ** d_out = nullptr;

    gpuErrchk(hipMalloc((void ***) &d_out, numBlocks*sizeof(CUDA_SPRING *)));
    gpuErrchk(hipMemcpy(d_out, sim->d_spring,
                         numBlocks*sizeof(CUDA_SPRING *),
                         hipMemcpyHostToDevice));

    reduceMinSpring(sim->springs.size(), THREADS_PER_BLOCK,
                    numBlocks, sim->d_spring, d_out);

    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(h_out, d_out, numBlocks*sizeof(CUDA_SPRING *), hipMemcpyDeviceToHost));

    leastStress = h_out[0];
    for (uint i = 1; i < numBlocks; i++) {
        leastStress = (leastStress->_max_stress < h_out[i]->_max_stress) ? leastStress : h_out[i];
    }

    qDebug() << "CUDA LEAST STRESS" << leastStress->_k;
}

// --------------------------------------------------------------------
void SimulatorParallel::removeNLeastStressedBars(int n) {
// --------------------------------------------------------------------
    //maxF_reducePass<THREADS_PER_BLOCK>(&structureMinStress, )

    // Test reduce
    //auto spring = thrust::reduce(sim->d_springs.begin(), sim->d_springs.end(), springPtr_minStressOperator);
    //qDebug() << "min spring" << spring;
    //thrust::device_vector<CUDA_SPRING *> d_springs_sorted(sim->d_springs);
    //thrust::sort(d_springs_sorted.begin(), d_springs_sorted.end());

    //qDebug() << d_springs_sorted.front();
}

// --------------------------------------------------------------------
void SimulatorParallel::reduceMinSpring(uint size, uint threads, uint blocks, CUDA_SPRING **d_spring, CUDA_SPRING **d_min_spring) {
// --------------------------------------------------------------------
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    int sharedMemSize = threads * sizeof(CUDA_SPRING *);

    warpReduce3MinSpring<<<dimGrid, dimBlock, sharedMemSize>>>(d_min_spring, d_spring, size);
}


// --------------------------------------------------------------------
//  GRAPHICS DATA OUTPUT
// --------------------------------------------------------------------


//  Registers CPU buffer ID to the GPU array for indices
//  NOTE: Assumes buffer input ID is bound
// --------------------------------------------------------------------
void SimulatorParallel::registerGraphicsVertexBuffer(uint cpuBufferId) {
// --------------------------------------------------------------------

    hipGraphicsGLRegisterBuffer(&CUDA_VERTEX_BUFFER, cpuBufferId, cudaGraphicsMapFlagsWriteDiscard);

}


//  Registers CPU buffer ID to the GPU array for indices
//  NOTE: Assumes buffer input ID is bound
// --------------------------------------------------------------------
void SimulatorParallel::registerGraphicsIndexBuffer(uint cpuBufferId) {
// --------------------------------------------------------------------

    hipGraphicsGLRegisterBuffer(&CUDA_INDEX_BUFFER, cpuBufferId, cudaGraphicsMapFlagsWriteDiscard);

}


//  Call GPU methods from the simulation to fill vertex buffer
// --------------------------------------------------------------------
void SimulatorParallel::exportGraphicsVertices() {
// --------------------------------------------------------------------

    float *vertices;
    size_t num_bytes;
    hipGraphicsMapResources(1, &CUDA_VERTEX_BUFFER);
    hipGraphicsResourceGetMappedPointer(reinterpret_cast<void **>(&vertices), &num_bytes, CUDA_VERTEX_BUFFER);
    sim->updateMassVertices(vertices);
    hipGraphicsUnmapResources(1, &CUDA_VERTEX_BUFFER);

}


//  Call GPU methods from the simulation to fill index buffer
// --------------------------------------------------------------------
void SimulatorParallel::exportGraphicsIndices() {
// --------------------------------------------------------------------

    unsigned int *indices;
    size_t num_bytes;
    hipGraphicsMapResources(1, &CUDA_INDEX_BUFFER);
    hipGraphicsResourceGetMappedPointer(reinterpret_cast<void **>(&indices), &num_bytes, CUDA_INDEX_BUFFER);
    sim->updateSpringIndices(indices);
    hipGraphicsUnmapResources(1, &CUDA_INDEX_BUFFER);

}

//  Unregisters GPU graphics resources
// --------------------------------------------------------------------
void SimulatorParallel::unregisterGraphicsResources() {
// --------------------------------------------------------------------

    hipGraphicsUnregisterResource(CUDA_VERTEX_BUFFER);
    hipGraphicsUnregisterResource(CUDA_INDEX_BUFFER);

}
